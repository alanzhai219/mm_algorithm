/*
 * Copyright 1993-2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation and 
 * any modifications thereto.  Any use, reproduction, disclosure, or distribution 
 * of this software and related documentation without an express license 
 * agreement from NVIDIA Corporation is strictly prohibited.
 * 
 */

/* Matrix multiplication: C = A * B.
 * Host code.
 *
 * This sample implements matrix multiplication and is exactly the same as
 * Chapter 7 of the programming guide.
 * It has been written for clarity of exposition to illustrate various CUDA
 * programming principles, not with the goal of providing the most
 * performant generic kernel for matrix multiplication.
 *
 * CUBLAS provides high-performance matrix multiplication.
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
//#include <cutil_inline.h>
//#include <helper_functions.h>

// includes, kernels
#include <hip/hip_runtime_api.h>
#include "matrixMul.h"
#include "matrixMul_tiling.cuh"

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest(int argc, char** argv);
void randomInit(float*, int);

////////////////////////////////////////////////////////////////////////////////
// Helper Functions
////////////////////////////////////////////////////////////////////////////////
#ifndef STRNCASECMP
#define STRNCASECMP strncasecmp
#endif

inline int stringRemoveDelimiter(char delimiter, const char *string) {
    int string_start = 0;

    while (string[string_start] == delimiter) {
        string_start++;
    }

    if (string_start >= (int)strlen(string)-1) {
        return 0;
    }

    return string_start;
}

inline bool checkCmdLineFlag(const int argc, const char **argv, const char *string_ref)
{
    bool bFound = false;

    if (argc >= 1)
    {
        for (int i=1; i < argc; i++)
        {
            int string_start = stringRemoveDelimiter('-', argv[i]);
            const char *string_argv = &argv[i][string_start];

            const char *equal_pos = strchr(string_argv, '=');
            int argv_length = (int)(equal_pos == 0 ? strlen(string_argv) : equal_pos - string_argv);

            int length = (int)strlen(string_ref);

            if (length == argv_length && !STRNCASECMP(string_argv, string_ref, length))
            {
                bFound = true;
                continue;
            }
        }
    }

    return bFound;
}

inline int getCmdLineArgumentInt(const int argc, const char **argv, const char *string_ref) {
    bool bFound = false;
    int value = -1;

    if (argc >= 1) {
        for (int i=1; i < argc; i++) {
            int string_start = stringRemoveDelimiter('-', argv[i]);
            const char *string_argv = &argv[i][string_start];
            int length = (int)strlen(string_ref);

            if (!STRNCASECMP(string_argv, string_ref, length)) {
                if (length+1 <= (int)strlen(string_argv)) {
                    int auto_inc = (string_argv[length] == '=') ? 1 : 0;
                    value = atoi(&string_argv[length + auto_inc]);
                } else {
                    value = 0;
                }

                bFound = true;
                continue;
            }
        }
    }

    if (bFound) {
        return value;
    } else {
        return 0;
    }
}


////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv) {
    runTest(argc, argv);

    exit(EXIT_SUCCESS);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void runTest(int argc, char** argv) {

    /****************************************************/
    /*  Preparations                                    */
    /****************************************************/

    printf("[Matrix Multiply Using CUDA] - Starting...\n");

    if (checkCmdLineFlag(argc, (const char **)argv, "help") ||
        checkCmdLineFlag(argc, (const char **)argv, "?")) {
        printf("Usage -device=n (n >= 0 for deviceID)\n");
        printf("      -wA=WidthA -hA=HeightA (Width x Height of Matrix A)\n");
        printf("      -wB=WidthB -hB=HeightB (Width x Height of Matrix B)\n");
        printf("  Note: Outer matrix dimensions of A & B matrices must be equal.\n");

        exit(EXIT_SUCCESS);
    }

    // By default, we use device 0, otherwise we override the device ID based on what is provided at the command line
    int devID = 0;

    if (checkCmdLineFlag(argc, (const char **)argv, "device")) {
        devID = getCmdLineArgumentInt(argc, (const char **)argv, "device");
        hipSetDevice(devID);
    }

    hipError_t error;
    hipDeviceProp_t deviceProp;
    error = hipGetDevice(&devID);

    if (error != hipSuccess) {
        printf("hipGetDevice returned error code %d, line(%d)\n", error, __LINE__);
    }

    error = hipGetDeviceProperties(&deviceProp, devID);

    if (deviceProp.computeMode == hipComputeModeProhibited) {
        fprintf(stderr, "Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
        exit(EXIT_SUCCESS);
    }

    if (error != hipSuccess) {
        printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
    } else {
        printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);
    }

    // utilities
    hipEvent_t start;
    hipEvent_t stop;
    float msecTotal;

    // set seed for rand()
    srand(2006);

    // allocate host memory for matrices A and B
    unsigned int size_A = WA * HA;
    unsigned int mem_size_A = sizeof(float) * size_A;
    float* h_A = (float*) malloc(mem_size_A);
    unsigned int size_B = WB * HB;
    unsigned int mem_size_B = sizeof(float) * size_B;
    float* h_B = (float*) malloc(mem_size_B);
    float flop = 2 * (float)WC * (float)HC * (float)WA;

    // initialize host memory
    randomInit(h_A, size_A);
    randomInit(h_B, size_B);
    
    // allocate device memory
    float* d_A;
    hipMalloc((void**) &d_A, mem_size_A);
    float* d_B;
    hipMalloc((void**) &d_B, mem_size_B);

    // allocate device memory for result
    unsigned int size_C = WC * HC;
    unsigned int mem_size_C = sizeof(float) * size_C;
    float* d_C;
    hipMalloc((void**) &d_C, mem_size_C);

    // allocate host memory for the result
    float* h_C = (float*) malloc(mem_size_C);

    /****************************************************/
    /*  Tiling without global mem coalescing            */
    /****************************************************/

    // create and start timer
    hipEventCreate(&start);
    hipEventRecord(start, NULL);
    // setup execution parameters
    auto threads = dim3(BLOCK_SIZE, BLOCK_SIZE);
    auto grid = dim3(WC / threads.x, HC / threads.y);
    // copy host memory to device
    hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice);
    // naive implementation
    matrixMul_tiling<<< grid, threads >>>(d_C, d_A, d_B, WA, WB);
    // copy result from device to host
    hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost);
    // stop and destroy timer
    hipEventCreate(&stop);
    hipEventRecord(stop, NULL);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&msecTotal, start, stop);
    printf("Tiling GPU\n");
    printf("Processing time: %f (ms), GFLOPS: %f \n", msecTotal, flop / msecTotal/ 1e+6);

    /****************************************************/
    /*  Cleaning                                        */
    /****************************************************/

    // clean up memory
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

// Allocates a matrix with random float entries.
void randomInit(float* data, int size) {
    for (int i = 0; i < size; ++i)
        data[i] = rand() / (float)RAND_MAX;
}
